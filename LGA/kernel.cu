#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "kernel.cuh"
#include<iostream>

__constant__ double t = 1;
__constant__ double tau = 2;


void run_kernels(Cell* matrix, int rows, int cols)//to wywolam z jakiegos cpp bo w cpp nie moge wywolac nic bezposrednio na karte graf cos jak getter w klasach
{
	dim3 blok = { 32,16,1 };//wymiar poj bloku ile watkow
	dim3 grid = { (cols + blok.x - 1) / blok.x, (rows + blok.y - 1) / blok.y,1 };//soatka watkow
	
	kernel_input << < grid, blok >> > (matrix, rows, cols);//uruchamiam kernela na siatce watkow ktorej wymiary sa opisane przez grid i przez blok
	hipDeviceSynchronize();
	kernel_output << < grid, blok >> > (matrix, rows, cols);
	hipDeviceSynchronize();
	
	 
}
__global__ void kernel_input(Cell* matrix, int rows, int cols)
{
	double w = 1.0 / 4.0;
	double feq[4] = { 0 };
	int index = threadIdx.x + blockDim.x * blockIdx.x + cols * (threadIdx.y + blockDim.y * blockIdx.y); //threadIdx- podaje wspolrzedne watku wew bloku, blockDim- wymiary bloku, blockIdx- podaje wspolrzedne bloku
	//index w ktorym el macierzy sie znjaduje
	if (matrix[index].wall == 1)
	{
		return;
	}
	for (int i = 0; i < 4; i++)
	{
		
		feq[i] = w * matrix[index].C;
		matrix[index].outTab[i] = matrix[index].inTab[i] + t / tau*(feq[i] - matrix[index].inTab[i]);
	}
}
__global__ void kernel_output(Cell* matrix, int rows, int cols)
{
	int index = threadIdx.x + blockDim.x * blockIdx.x + cols * (threadIdx.y + blockDim.y * blockIdx.y); //threadIdx- podaje wspolrzedne watku wew bloku, blockDim- wymiary bloku, blockIdx- podaje wspolrzedne bloku
	if (matrix[index].wall == 0)
	{
		if (matrix[index - cols].wall == 0)
			matrix[index].inTab[0] = matrix[index - cols].outTab[0];//w dol
		else
			matrix[index].inTab[0] = matrix[index].outTab[1];//w dol

		if (matrix[index + cols].wall ==0)
			matrix[index].inTab[1] = matrix[index + cols].outTab[1];
		else
			matrix[index].inTab[1] = matrix[index].outTab[0];//w dol

		if (matrix[index -1].wall == 0)
			matrix[index].inTab[2] = matrix[index - 1].outTab[2];//w prawo
		else
			matrix[index].inTab[2] = matrix[index].outTab[3];//w dol

		if (matrix[index + 1].wall == 0)
			matrix[index].inTab[3] = matrix[index + 1].outTab[3];
		else
			matrix[index].inTab[3] = matrix[index].outTab[2];//w dol


		matrix[index].C = matrix[index].inTab[0] + matrix[index].inTab[1] + matrix[index].inTab[2] + matrix[index].inTab[3];
	}
}

__global__ void kernel_draw(Cell* matrix, int rows, int cols, float* vbo)
{
	int index = threadIdx.x + blockDim.x * blockIdx.x + cols * (threadIdx.y + blockDim.y * blockIdx.y); //threadIdx- podaje wspolrzedne watku wew bloku, blockDim- wymiary bloku, blockIdx- podaje wspolrzedne bloku

	if (matrix[index].wall == 1)
	{
		vbo[((index) * 5) + 2] = 1.0f;
		vbo[((index) * 5) + 3] = 0;
		vbo[((index) * 5) + 4] = 0;
		return;
	}
	
	vbo[((index) * 5) + 2] =  matrix[index].C;
	vbo[((index) * 5) + 3] = matrix[index].C;
	vbo[((index) * 5) + 4] = matrix[index].C;
	
}


void draw(Cell* matrix, int rows, int cols, float* vbo)
{
	dim3 blok = { 32, 16, 1 };
	dim3 grid = { (cols + blok.x - 1) / blok.x, (rows + blok.y - 1) / blok.y,1 };
	kernel_draw <<< grid, blok >>> (matrix, rows, cols, vbo);
	hipDeviceSynchronize();
}